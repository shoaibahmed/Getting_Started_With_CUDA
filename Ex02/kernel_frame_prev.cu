
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Yourkernel(uchar3 *dary,float t,int DIMX,int DIMY)
{
	/* Insert your kernel here */
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int offset = blockId * blockDim.x + threadId;
  uchar3 color;
  if ((threadId < (DIMX / 2)) && (blockId < (DIMY / 2)))
    color = make_uchar3(255,0,0);
  else if ((threadId > (DIMX / 2)) && (blockId < (DIMY / 2)))
    color = make_uchar3(0,255,0);
  else if ((threadId < (DIMX / 2)) && (blockId > (DIMY / 2)))
    color = make_uchar3(255,255,0);
  else
    color = make_uchar3(0,0,255);
  dary[offset] = color;
}

void simulate(uchar3 *ptr, int tick, int w, int h)
{
	/* ptr is a pointer to an array of size w*h*sizeof(uchar3).
	   uchar3 is a structure with x,y,z coordinates to contain
	   red,yellow,blue - values for a pixel (Range [0,255])
	*/
	hipError_t err=hipSuccess;
	hipEvent_t start,stop;
	float elapsedtime;

	hipEventCreate  ( &start);
	hipEventCreate  ( &stop);

	hipEventRecord(start);
	/* Space for
	Yourkernel
	*/
  Yourkernel<<<h,w>>>(ptr, tick, w, h);
	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n",
				 hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime, start, stop);
	printf("Time used: %.1f (ms)\n",elapsedtime);

	hipEventDestroy  ( start);
	hipEventDestroy  ( stop);

	printf("Please type ESC in graphics and afterwards RETURN in cmd-screen to finish\n");
}
