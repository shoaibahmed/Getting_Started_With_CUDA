
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#define MAX_THREADS_PER_BLOCK 1024

__global__ void Yourkernel(uchar3 *dary,float t,int DIMX,int DIMY)
{
	/* Insert your kernel here */
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = (i * DIMX) + (j);
	
	uchar3 color;
	// color = make_uchar3(((float)i / DIMX) * 256, ((float)j / DIMY) * 256 , 0);
	
	// Distinct color for each block
	// color = make_uchar3(((float)i / DIMX) * 256, ((float)j / DIMY) * 256, 0);
	color = make_uchar3(((float)blockIdx.x / blockDim.x) * 256, ((float)blockIdx.y / blockDim.x) * 256 , 0);
	assert(offset < (DIMX * DIMY));
	dary[offset] = color;
}

void simulate(uchar3 *ptr, int tick, int w, int h)
{
	/* ptr is a pointer to an array of size w*h*sizeof(uchar3).
	   uchar3 is a structure with x,y,z coordinates to contain
	   red,yellow,blue - values for a pixel (Range [0,255])
	*/
	hipError_t err=hipSuccess;
	hipEvent_t start,stop;
	float elapsedtime;

	hipEventCreate  ( &start);
	hipEventCreate  ( &stop);

	hipEventRecord(start);

	/* Space for
	Yourkernel
	*/
	int divisions = 3; // 9 blocks
	dim3 dimBlock(32, 32);
	// dim3 dimBlock((int)(w / divisions), (int)(h / divisions));
	dim3 dimGrid((w + dimBlock.x - 1) / dimBlock.x, (h + dimBlock.y - 1) / dimBlock.y);
	Yourkernel<<<dimGrid, dimBlock>>>(ptr, tick, w, h);

	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime, start, stop);
	printf("Time used: %.1f (ms)\n",elapsedtime);

	hipEventDestroy  ( start);
	hipEventDestroy  ( stop);

	printf("Please type ESC in graphics and afterwards RETURN in cmd-screen to finish\n");
}
