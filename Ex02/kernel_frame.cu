
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define MAX_THREADS_PER_BLOCK 1024
#define DRAW_GRADIENT_MAP true

__global__ void ColorBufferFillKernel(uchar3 *dary, float t, int DIMX, int DIMY, int numBlocksWithSameColor)
{
	/* Insert your kernel here */
	int i = blockIdx.x * blockDim.x + threadIdx.x; 
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = (i * DIMX) + (j);

	uchar3 color;
	
#if DRAW_GRADIENT_MAP
	// color = make_uchar3(((float)i / DIMX) * 256, ((float)j / DIMY) * 256 , 0);
	
	// Distinct color for each block
	// color = make_uchar3(((float)i / DIMX) * 256, ((float)j / DIMY) * 256, 0);
	// color = make_uchar3(((float)blockIdx.x / gridDim.x) * 255, ((float)blockIdx.y / gridDim.y) * 255, 0);

	int blockColorIdxX = blockIdx.x / numBlocksWithSameColor + 1;
	int normalizerX = gridDim.x / numBlocksWithSameColor + 1;
	float xProportion = (float)((blockIdx.x % numBlocksWithSameColor) * blockDim.x + threadIdx.x) / (numBlocksWithSameColor * blockDim.x);
	int blockColorIdxY = blockIdx.y / numBlocksWithSameColor + 1;
	int normalizerY = gridDim.y / numBlocksWithSameColor + 1;
	float yProportion = (float)((blockIdx.y % numBlocksWithSameColor) * blockDim.y + threadIdx.y) / (numBlocksWithSameColor * blockDim.y);

	int currentBlockColorX = (((float)blockColorIdxX / normalizerX) * 255);
	int currentBlockColorY = (((float)blockColorIdxY / normalizerY) * 255);

	// Get last block colors
	int lastBlockXColor = 0, lastBlockYColor = 0;
	if (blockColorIdxX > 0)
		lastBlockXColor = ((float)(blockColorIdxX - 1) / normalizerX) * 255;
	if (blockColorIdxY > 0)
		lastBlockYColor = ((float)(blockColorIdxY - 1) / normalizerX) * 255;

	// color = make_uchar3(((float)blockColorIdxX / normalizerX) * 255, ((float)blockColorIdxY / normalizerY) * 255, 0);
	color = make_uchar3((xProportion) * currentBlockColorX + (1.0 - xProportion) * lastBlockXColor, 
		(yProportion) * currentBlockColorY + (1.0 - yProportion) * lastBlockYColor, 0);

	dary[offset] = color;
#else
	int blockColorIdxX = blockIdx.x / numBlocksWithSameColor;
	int normalizerX = gridDim.x / numBlocksWithSameColor;
	int blockColorIdxY = blockIdx.y / numBlocksWithSameColor;
	int normalizerY = gridDim.y / numBlocksWithSameColor;
	color = make_uchar3(((float)blockColorIdxX / normalizerX) * 255, ((float)blockColorIdxY / normalizerY) * 255, 0);
	dary[offset] = color;
#endif
}

void simulate(uchar3 *ptr, int tick, int w, int h)
{
	/* ptr is a pointer to an array of size w*h*sizeof(uchar3).
	   uchar3 is a structure with x,y,z coordinates to contain
	   red,yellow,blue - values for a pixel (Range [0,255])
	*/
	hipError_t err=hipSuccess;
	hipEvent_t start,stop;
	float elapsedtime;

	hipEventCreate  ( &start);
	hipEventCreate  ( &stop);

	hipEventRecord(start);

	/* Space for
	Yourkernel
	*/
	int divisions = 3; // 9 blocks
	int blockDim = 25;

	// Pick the ideal dimensions of kernel

	dim3 dimBlock(blockDim, blockDim);
	// dim3 dimBlock((int)(w / divisions), (int)(h / divisions));
	dim3 dimGrid((w + dimBlock.x - 1) / dimBlock.x, (h + dimBlock.y - 1) / dimBlock.y);
	printf("Grid dims: (%d, %d)\n", dimGrid.x, dimGrid.y);

	// Determine the number of kernels to be colored the same
	int numBlocksWithSameColor = floor(h / (divisions * blockDim));
	printf("Number of blocks with same color: %d\n", numBlocksWithSameColor);
	
	// Start the kernel
	ColorBufferFillKernel<<<dimGrid, dimBlock>>>(ptr, tick, w, h, numBlocksWithSameColor);

	err=hipGetLastError();
	if(err!=hipSuccess) {
		fprintf(stderr,"Error executing the kernel - %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime, start, stop);
	printf("Time used: %.1f (ms)\n",elapsedtime);

	hipEventDestroy  ( start);
	hipEventDestroy  ( stop);

	printf("Please type ESC in graphics and afterwards RETURN in cmd-screen to finish\n");
}
