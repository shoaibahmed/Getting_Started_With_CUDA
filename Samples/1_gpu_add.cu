
#include <hip/hip_runtime.h>
#include "stdio.h"

// Kernel addition on GPU
__global__ void add(int a, int* b)
{
	*b += a * 100;
}

// Main function on the host
int main()
{
	int b, *dev_b;
	hipMalloc((void **) &dev_b, sizeof(int));
	add <<< 1, 1 >>> (2, dev_b);
	hipMemcpy(&b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_b);
	printf ("B: %d\n", b);
	return 0;
}