#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 1000000
#define BLOCK_DIM 1000
#define ARRAY_SIZE 10

#define USE_ATOMICS true

__global__ void naiveAddKernel(float* d_arr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	idx = idx % ARRAY_SIZE;
	d_arr[idx] += 1;
}

__global__ void atomicAddKernel(float* d_arr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	idx = idx % ARRAY_SIZE;
	atomicAdd(&d_arr[idx], 1);
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Initialize the array
	float *d_arr, *h_arr;
	hipMalloc((void **) &d_arr, ARRAY_SIZE * sizeof(float));
	hipMemset((void **) &d_arr, 0, sizeof(float));

	hipEventRecord(start);
	#if USE_ATOMICS
		atomicAddKernel<<<NUM_THREADS / BLOCK_DIM, BLOCK_DIM>>>(d_arr);
	#else
		naiveAddKernel<<<NUM_THREADS / BLOCK_DIM, BLOCK_DIM>>>(d_arr);
	#endif
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Kernel execution time: %f ms\n", milliseconds);

	// Copy back the results
	hipMemcpy(h_arr, d_arr, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f", h_arr[i]);
		printf ((i % 4 == 3)? "\n" : "\t");
	}

	hipFree(h_arr);
	hipFree(d_arr);

	return 0;
}