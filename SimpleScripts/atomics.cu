#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 1000000
#define BLOCK_DIM 1000
#define ARRAY_SIZE 10

#define USE_ATOMICS true

__global__ void naiveAddKernel(float* d_arr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	idx = idx % ARRAY_SIZE;
	d_arr[idx] += 1;
}

__global__ void atomicAddKernel(float* d_arr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	idx = idx % ARRAY_SIZE;
	atomicAdd(&d_arr[idx], 1);
}

int main()
{
	// Initialize the array
	float *d_arr, *h_arr;
	hipMalloc((void **) &d_arr, ARRAY_SIZE * sizeof(float));
	hipMemset((void **) &d_arr, 0, sizeof(float));

	#if USE_ATOMICS
		atomicAddKernel<<<NUM_THREADS / BLOCK_DIM, BLOCK_DIM>>>(d_arr);
	#else
		naiveAddKernel<<<NUM_THREADS / BLOCK_DIM, BLOCK_DIM>>>(d_arr);
	#endif

	// Copy back the results
	hipMemcpy(h_arr, d_arr, ARRAY_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		printf("%f\t", h_arr[i]);
	}

	hipFree(d_arr);

	return 0;
}